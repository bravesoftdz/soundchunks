/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
/*   File:         seq_main.c   (an sequential version)                      */
/*   Description:  This program shows an example on how to call a subroutine */
/*                 that implements a simple k-means clustering algorithm     */
/*                 based on Euclid distance.                                 */
/*   Input file format:                                                      */
/*                 ascii  file: each line contains 1 data object             */
/*                 binary file: first 4-byte integer is the number of data   */
/*                 objects and 2nd integer is the no. of features (or        */
/*                 coordinates) of each object                               */
/*                                                                           */
/*   Author:  Wei-keng Liao                                                  */
/*            ECE Department Northwestern University                         */
/*            email: wkliao@ece.northwestern.edu                             */
/*   Copyright, 2005, Wei-keng Liao                                          */
/*                                                                           */
/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */

// Copyright (c) 2005 Wei-keng Liao
// Copyright (c) 2011 Serban Giuroiu
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.

// -----------------------------------------------------------------------------

#include <stdio.h>
#include <stdlib.h>
#include <string.h>     /* strtok() */
#include <sys/types.h>  /* open() */
#include <sys/stat.h>
#include <fcntl.h>
#include "unistd.h"     /* getopt() */
#include "kmeans.h"

int      _debug;

#define malloc2D(name, xDim, yDim, type) do {               \
    name = (type **)malloc(xDim * sizeof(type *));          \
    assert(name != NULL);                                   \
    name[0] = (type *)malloc(xDim * yDim * sizeof(type));   \
    assert(name[0] != NULL);                                \
    for (size_t i = 1; i < xDim; i++)                       \
        name[i] = name[i-1] + yDim;                         \
} while (0)

float** file_read(int, char*, int*, int*);
int     file_write(char*, int, int, int, float**, int*);

/*---< usage() >------------------------------------------------------------*/
static void usage(char *argv0, float threshold) {
    const char *help =
        "Usage: %s [switches] -i filename -n num_clusters\n"
        "       -i filename    : file containing data to be clustered\n"
        "       -c filename    : file containing init centroids\n"
        "       -b             : input file is in binary format (default no)\n"
        "       -n num_clusters: number of clusters (K must > 1)\n"
        "       -t threshold   : threshold value (default %.4f)\n"
        "       -d             : enable debug mode\n";
    fprintf(stderr, help, argv0, threshold);
    exit(-1);
}

typedef float real_t;

/*---< main() >-------------------------------------------------------------*/
int main(int argc, char **argv) {
           int     opt;
    extern char   *optarg;
    extern int     optind;
           int     isBinaryFile;

           int     numClusters, numCoords, numObjs;
           int    *membership;    /* [numObjs] */
           char   *filename, *centFname;
           float **objects;       /* [numObjs][numCoords] data objects */
           float **clusters;      /* [numClusters][numCoords] cluster center */
           float   threshold;

    /* some default values */
    _debug           = 0;
    threshold        = 0.001;
    numClusters      = 0;
    isBinaryFile     = 0;
    filename         = NULL;
    centFname        = NULL;

    setbuf(stdout, NULL);
    setbuf(stderr, NULL);
    
    while ( (opt=getopt(argc,argv,"p:i:c:n:t:abdo"))!= EOF) {
        switch (opt) {
            case 'i': filename=optarg;
                      break;
            case 'c': centFname=optarg;
                      break;
            case 'b': isBinaryFile = 1;
                      break;
            case 't': threshold=atof(optarg);
                      break;
            case 'n': numClusters = atoi(optarg);
                      break;
            case 'd': _debug = 1;
                      break;
            case '?': usage(argv[0], threshold);
                      break;
            default: usage(argv[0], threshold);
                      break;
        }
    }

    if (filename == 0 || numClusters <= 1) usage(argv[0], threshold);

    /* read data points from file ------------------------------------------*/
    objects = file_read(isBinaryFile, filename, &numObjs, &numCoords);
    if (objects == NULL) exit(1);

    if (centFname != 0)
    {
      int yc, xc;
      clusters = file_read(isBinaryFile, centFname, &yc, &xc);
      if (yc != numClusters || xc != numCoords)
      {
        printf("Cendroids mismatch: numCoords %d->%d numClusters %d->%d\n", numCoords, xc, numClusters, yc);
      }
    }
    else
    {
      clusters = NULL;
    }

    /* start the timer for the core computation -----------------------------*/
    /* membership: the cluster id for each data object */
    membership = (int*) malloc(numObjs * sizeof(int));
    assert(membership != NULL);

    for (int i = 0; i < numObjs; i++) {
        membership[i] = i;
    }

    thrust::device_vector<real_t> *data;
    thrust::device_vector<int> *labels;
    thrust::device_vector<real_t> *centroids;
    thrust::device_vector<real_t> *distances;
    
    hipSetDevice(0);
    data = new thrust::device_vector<real_t>(numObjs*numCoords);
    labels = new thrust::device_vector<int>(numObjs);
    centroids = new thrust::device_vector<real_t>(numClusters * numCoords);
    distances = new thrust::device_vector<real_t>(numObjs);
    
    thrust::copy(&(objects[0][0]), &(objects[numObjs-1][numCoords-1]), data->begin());

    if (clusters != 0)
    {
      thrust::copy(&(clusters[0][0]), &(clusters[numClusters-1][numCoords-1]), centroids->begin());
    }

    thrust::copy(&(membership[0]), &(membership[numObjs-1]), labels->begin());
    
    printf("numObjs %d numCoords %d numClusters %d\n", numObjs, numCoords, numClusters);
      
    kmeans::kmeans<real_t>(numObjs, numCoords, numClusters, &data, &labels, &centroids, &distances, 1, INT_MAX, clusters == 0, threshold);

    malloc2D(clusters, numClusters, numCoords, float);
    thrust::copy(centroids->begin(), centroids->end(), &(clusters[0][0]));
    
    thrust::copy(labels->begin(), labels->end(), &(membership[0]));
    
    free(objects[0]);
    free(objects);

    /* output: the coordinates of the cluster centres ----------------------*/
    file_write(filename, numClusters, numObjs, numCoords, clusters,
               membership);

    free(membership);
    free(clusters[0]);
    free(clusters);

    return(0);
}

