#include <thrust/device_vector.h>
#include "timer.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include <cstdlib>
#include "kmeans.h"

template<typename T>
void fill_array(T& array, int m, int n) {
  for(int i = 0; i < m; i++) {
    for(int j = 0; j < n; j++) {
      array[i * n + j] = (i % 2)*3 + j;
    }
  }
}

template<typename T>
void random_data(thrust::device_vector<T>& array, int n, int d, int k) {
  thrust::host_vector<T> host_array(n*d);
  for(int i = 0; i < n; i++) {
  for(int j = 0; j < d; j++) {
    //    host_array[i] = (T)rand()/(T)RAND_MAX;
    host_array[i*d+j] = i%k;
    //    host_array[j*n+i] = i%k;
  }
  }
  array = host_array;
}

void random_labels(thrust::device_vector<int>& labels, int n, int k) {
  thrust::host_vector<int> host_labels(n);
  for(int i = 0; i < n; i++) {
    host_labels[i] = rand() % k;
  }
  labels = host_labels;
}

typedef float real_t;

int main() {
  int max_iterations = 10000;
  int n = 260753;
  //  int d = 298;
  //int k = 100;
  int d = 3;
  int k = 10;
  double thresh = 1e-3;

  int n_gpu;
  hipGetDeviceCount(&n_gpu);
  n_gpu=1;

  std::cout << n_gpu << " gpus." << std::endl;

  thrust::device_vector<real_t> *data[16];
  thrust::device_vector<int> *labels[16];
  thrust::device_vector<real_t> *centroids[16];
  thrust::device_vector<real_t> *distances[16];
  for (int q = 0; q < n_gpu; q++) {
    hipSetDevice(q);
    data[q] = new thrust::device_vector<real_t>(n/n_gpu*d);
    labels[q] = new thrust::device_vector<int>(n/n_gpu*d);
    centroids[q] = new thrust::device_vector<real_t>(k * d);
    distances[q] = new thrust::device_vector<real_t>(n);
  }

  std::cout << "Generating random data" << std::endl;
  std::cout << "Number of points: " << n << std::endl;
  std::cout << "Number of dimensions: " << d << std::endl;
  std::cout << "Number of clusters: " << k << std::endl;
  std::cout << "Max. number of iterations: " << max_iterations << std::endl;
  std::cout << "Stopping threshold: " << thresh << std::endl;

  /* Intializes random number generator */
  //srand((unsigned) time(&t));
  srand(777);
  
  for (int q = 0; q < n_gpu; q++) {
    random_data<real_t>(*data[q], n/n_gpu, d, k);
    random_labels(*labels[q], n/n_gpu, k);
  }
  kmeans::timer t;
  t.start();
  kmeans::kmeans<real_t>(n, d, k, data, labels, centroids, distances, n_gpu, max_iterations, true, thresh);
  float time = t.stop();
  std::cout << "  Time: " << time/1000.0 << " s" << std::endl;

  // debug
  int printcenters=1;
  if(printcenters){
    thrust::host_vector<real_t> *ctr = new thrust::host_vector<real_t>(*centroids[0]);
    for(unsigned int ii=0;ii<k;ii++){
      fprintf(stderr,"ii=%d of k=%d ",ii,k);
      for(unsigned int jj=0;jj<d;jj++){
        fprintf(stderr,"%g ",(*ctr)[d*ii+jj]);
      }
      fprintf(stderr,"\n");
      fflush(stderr);
    }
  }
  
  for (int q = 0; q < n_gpu; q++) {
    delete(data[q]);
    delete(labels[q]);
    delete(centroids[q]);
    delete(distances[q]);
  }
}
